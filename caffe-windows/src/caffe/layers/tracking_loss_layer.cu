#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/tracking_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void TrackingLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[1]->gpu_data(),  // b
      diff_relative_.mutable_gpu_data());  // a_i-b_i
  caffe_gpu_powx(
      count,
      diff_relative_.mutable_gpu_data(),  // a_i-b_i
      Dtype(2),
      diff_relative_sq_.mutable_gpu_data());  // (a_i-b_i)^2
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_relative_sq_.gpu_data(),  // (a_i-b_i)^2
      summer_vec_relative_.gpu_data(),
      Dtype(0.0),
      dist_relative_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2
  Dtype margin = this->layer_param_.contrastive_loss_param().margin();
  bool legacy_version =
      this->layer_param_.contrastive_loss_param().legacy_version();
  Dtype loss(0.0);
  for (int i = 0; i < bottom[0]->num(); ++i) {
    if (static_cast<int>(bottom[2]->cpu_data()[i])) {  // similar pairs
      loss += dist_relative_sq_.cpu_data()[i];
    } else {  // dissimilar pairs
      loss += std::max(margin - diff_relative_.cpu_data()[i], Dtype(0.0));// change by YangXS: dist_relative_sq_ to diff_relative_
    }
  }
  loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
  
  //change by GaoJY, adding euclidean loss///////////////////////////////////////////////
  int count_detect = bottom[3]->count();
  Dtype lambda1 = this->layer_param_.tracking_loss_param().lambda1();
  caffe_gpu_sub(
      count_detect,
      bottom[3]->gpu_data(),
      bottom[5]->gpu_data(),
      diff_dectect_.mutable_gpu_data());
  Dtype dot;
  caffe_gpu_dot(count_detect, diff_dectect_.gpu_data(), diff_dectect_.gpu_data(), &dot);
  
  int count_detect_1 = bottom[4]->count();
  caffe_gpu_sub(
      count_detect_1,
      bottom[4]->gpu_data(),
      bottom[6]->gpu_data(),
      diff_dectect_1_.mutable_gpu_data());
  Dtype dot1;
  caffe_gpu_dot(count_detect_1, diff_dectect_1_.gpu_data(), diff_dectect_1_.gpu_data(), &dot1);
  
  loss = loss + lambda1 * (dot + dot1) / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
  //change by GaoJY, adding euclidean loss end///////////////////////////////////////////////
  
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void CLLBackward(const int count, const int channels,
    const Dtype margin, const bool legacy_version, const Dtype alpha,
    const Dtype* y, const Dtype* diff_relative_, const Dtype* dist_relative_sq_,
    Dtype *bottom_diff) {
  CUDA_KERNEL_LOOP(i, count) {
    int n = i / channels;  // the num index, to access y and dist_sq
    if (static_cast<int>(y[n])) {  // similar pairs
      bottom_diff[i] = alpha * diff_relative_[i];
    } else {  // dissimilar pairs
      // change by YangXS:
      if ((margin - diff_relative_[n]) > 0.0) {
        bottom_diff[i] = -1/2.0 * alpha;
      } else {
        bottom_diff[i] = 0;
      }
    }
  }
}

template <typename Dtype>
void TrackingLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const int count = bottom[0]->count();
      const int channels = bottom[0]->channels();
      Dtype margin = this->layer_param_.contrastive_loss_param().margin();
      const bool legacy_version =
          this->layer_param_.contrastive_loss_param().legacy_version();
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha_relative = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(bottom[0]->num());
      // NOLINT_NEXT_LINE(whitespace/operators)
      CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, channels, margin, legacy_version, alpha_relative,
          bottom[2]->gpu_data(),  // pair similarity 0 or 1
          diff_relative_.gpu_data(),  // the cached eltwise difference between a and b
          dist_relative_sq_.gpu_data(),  // the cached square distance between a and b
          bottom[i]->mutable_gpu_diff());
      CUDA_POST_KERNEL_CHECK;
    }
  }
  
  //change by GaoJY, adding euclidean loss///////////////////////////////////////////////
  Dtype lambda1 = this->layer_param_.tracking_loss_param().lambda1();
  for (int i = 3; i < 5; ++i) {
    if (propagate_down[i] && i == 3) {
      const Dtype alpha = lambda1 * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          diff_dectect_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
    }
	
	if (propagate_down[i] && i == 4) {
      const Dtype alpha_1 = lambda1 * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha_1,                              // alpha
          diff_dectect_1_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
    }
  }
  //change by GaoJY, adding euclidean loss end///////////////////////////////////////////////
}

INSTANTIATE_LAYER_GPU_FUNCS(TrackingLossLayer);

}  // namespace caffe
